#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <stdlib.h>
#include <getopt.h>
#include <memory.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#include "gdrapi.h"
#include "common.hpp"

using namespace gdrcopy::test;

// Measuring the ping-pong latency of flag only.
__global__ void pp_kernel(uint32_t *gpu_flag_buf, uint32_t *cpu_flag_buf, uint32_t num_iters)
{
    uint32_t i = 1;
    WRITE_ONCE(*cpu_flag_buf, i);
    __threadfence_system();
    while (i < num_iters) {
        uint32_t val;
        do {
            val = READ_ONCE(*gpu_flag_buf);
        }
        while (val != i);

        ++val;
        WRITE_ONCE(*cpu_flag_buf, val);

        i = val;
    }
}

// This kernel emulates data + flag model. We consume the data by copying it to another GPU buffer.
__global__ void pp_data_kernel(uint32_t *gpu_flag_buf, uint32_t *cpu_flag_buf, uint32_t num_iters, uint32_t *A, uint32_t *B, size_t data_size)
{
    uint64_t my_tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t num_threads = gridDim.x * blockDim.x;
    uint64_t num_elements = data_size / sizeof(*A);
    uint32_t data_val;
    uint32_t flag_val;
    uint32_t i = 1;

    if (threadIdx.x == 0) {
        WRITE_ONCE(cpu_flag_buf[blockIdx.x], i);
        __threadfence_system();
    }
    __syncthreads();

    for (; i < num_iters; ++i) {
        if (threadIdx.x == 0) {
            do {
                flag_val = READ_ONCE(*gpu_flag_buf);
            }
            while (flag_val != i);
            __threadfence_system();
        }
        __syncthreads();

        for (uint64_t idx = my_tid; idx < num_elements; idx += num_threads)
            B[idx] = A[idx];
        __syncthreads();

        if (threadIdx.x == 0) {
            ++flag_val;
            WRITE_ONCE(cpu_flag_buf[blockIdx.x], flag_val);
            __threadfence_system();
        }
    }
}

static int dev_id = 0;
static uint32_t num_iters = 1000;
static size_t data_size = 0;

static unsigned int num_blocks = 8;
static unsigned int num_threads_per_block = 1024;

static unsigned int timeout = 10;  // in s
// Counter value before checking timeout.
static unsigned long int timeout_check_threshold = 1000000UL;
static unsigned long int timeout_counter = 0;

static void print_usage(const char *path)
{
    cout << "Usage: " << path << " [options]" << endl;
    cout << endl;
    cout << "Options:" << endl;
    cout << "   -h              Print this help text" << endl;
    cout << "   -d <gpu>        GPU ID (default: " << dev_id << ")" << endl;
    cout << "   -t <iters>      Number of iterations (default: " << num_iters << ")" << endl;
    cout << "   -u <timeout>    Timeout in second. 0 to disable. (default: " << timeout << ")" << endl;
    cout << "   -a <fn>         GPU buffer allocation function (default: hipMalloc)" << endl;
    cout << "                       Choices: hipMalloc, hipMemCreate" << endl;
    cout << "   -s <size>       Data size (default: " << data_size << ")" << endl;
    cout << "                       0 means measuring the visibility latency of the flag" << endl;
    cout << "   -B <nblocks>    Number of CUDA blocks (default: " << num_blocks << ")" << endl;
    cout << "   -T <nthreads>   Number of threads per CUDA blocks (default: " << num_threads_per_block << ")" << endl;
}

/**
 * Return time difference in us.
 */
static inline double time_diff(struct timespec start, struct timespec end)
{
    return (double)((end.tv_nsec - start.tv_nsec) / 1000.0 + (end.tv_sec - start.tv_sec) * 1000000.0);
}

static inline void check_timeout(struct timespec start, double timeout_us)
{
    hipError_t status;
    const char *cu_status_name;
    struct timespec now;
    double time_used_us;
    if (timeout_us > 0) {
        ++timeout_counter;
        if (timeout_counter >= timeout_check_threshold) {
            clock_gettime(MYCLOCK, &now);
            time_used_us = time_diff(start, now);
            if (time_used_us > timeout_us) {
                cerr << "ERROR: TIMEOUT!!!" << endl;
                status = hipStreamQuery(0);
                hipDrvGetErrorName(status, &cu_status_name);
                cerr << "hipStreamQuery(0) returned " << cu_status_name << endl;
                abort();
            }
            timeout_counter = 0;
        }
    }
}

int main(int argc, char *argv[])
{
    uint32_t *g_gpu_flag_buf = NULL;
    uint32_t *h_cpu_flag_buf = NULL;

    hipDeviceptr_t d_gpu_flag_buf;
    hipDeviceptr_t d_cpu_flag_buf;

    gpu_mem_handle_t gpu_flag_mhandle;

    uint32_t *g_A;

    hipDeviceptr_t d_A = 0;
    hipDeviceptr_t d_B = 0;

    gpu_mem_handle_t A_mhandle;
    gpu_mem_handle_t B_mhandle;

    size_t data_buffer_size = 0;

    uint32_t *init_buf = NULL;

    struct timespec beg, end;
    double lat_us;
    double timeout_us;

    gpu_memalloc_fn_t galloc_fn = gpu_mem_alloc;
    gpu_memfree_fn_t gfree_fn = gpu_mem_free;

    while(1) {
        int c;
        c = getopt(argc, argv, "d:t:u:a:s:B:T:h");
        if (c == -1)
            break;

        switch (c) {
            case 'd':
                dev_id = strtol(optarg, NULL, 0);
                break;
            case 't':
                num_iters = strtol(optarg, NULL, 0);
                break;
            case 'u':
                timeout = strtol(optarg, NULL, 0);
                break;
            case 'a':
                if (strcmp(optarg, "hipMalloc") == 0) {
                    galloc_fn = gpu_mem_alloc;
                    gfree_fn = gpu_mem_free;
                }
                else if (strcmp(optarg, "hipMemCreate") == 0) {
                    galloc_fn = gpu_vmm_alloc;
                    gfree_fn = gpu_vmm_free;
                }
                else {
                    cerr << "Unrecognized fn argument" << endl;
                    exit(EXIT_FAILURE);
                }
                break;
            case 's':
                data_size = strtol(optarg, NULL, 0);
                break;
            case 'B':
                num_blocks = strtol(optarg, NULL, 0);
                break;
            case 'T':
                num_threads_per_block = strtol(optarg, NULL, 0);
                break;
            case 'h':
                print_usage(argv[0]);
                exit(EXIT_SUCCESS);
            default:
                cerr << "ERROR: invalid option" << endl;
                exit(EXIT_FAILURE);
        }
    }

    const bool do_consume_data = (data_size > 0);

    if (data_size % sizeof(*g_A) != 0) {
        cerr << "ERROR: data_size must be divisible by " << sizeof(*g_A) << "." << endl;
        exit(EXIT_FAILURE);
    }

    if (num_blocks <= 0) {
        cerr << "ERROR: nblocks must be at least 1." << endl;
        exit(EXIT_FAILURE);
    }

    if (num_threads_per_block <= 0) {
        cerr << "ERROR: nthreads must be at least 1." << endl;
        exit(EXIT_FAILURE);
    }

    timeout_us = timeout * 1000000.0;

    ASSERTDRV(hipInit(0));

    int n_devices = 0;
    ASSERTDRV(hipGetDeviceCount(&n_devices));

    hipDevice_t dev;
    for (int n = 0; n < n_devices; ++n) {

        char dev_name[256];
        int dev_pci_domain_id;
        int dev_pci_bus_id;
        int dev_pci_device_id;

        ASSERTDRV(hipDeviceGet(&dev, n));
        ASSERTDRV(hipDeviceGetName(dev_name, sizeof(dev_name) / sizeof(dev_name[0]), dev));
        ASSERTDRV(hipDeviceGetAttribute(&dev_pci_domain_id, hipDeviceAttributePciDomainId, dev));
        ASSERTDRV(hipDeviceGetAttribute(&dev_pci_bus_id, hipDeviceAttributePciBusId, dev));
        ASSERTDRV(hipDeviceGetAttribute(&dev_pci_device_id, hipDeviceAttributePciDeviceId, dev));

        cout  << "GPU id:" << n << "; name: " << dev_name
              << "; Bus id: "
              << std::hex
              << std::setfill('0') << std::setw(4) << dev_pci_domain_id
              << ":" << std::setfill('0') << std::setw(2) << dev_pci_bus_id
              << ":" << std::setfill('0') << std::setw(2) << dev_pci_device_id
              << std::dec
              << endl;
    }
    cout << "selecting device " << dev_id << endl;
    ASSERTDRV(hipDeviceGet(&dev, dev_id));

    hipCtx_t dev_ctx;
    ASSERTDRV(hipDevicePrimaryCtxRetain(&dev_ctx, dev));
    ASSERTDRV(hipCtxSetCurrent(dev_ctx));

    int max_threads_per_block;
    ASSERTDRV(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, dev));

    if (num_threads_per_block > max_threads_per_block) {
        cerr << "ERROR: nthreads can be at most " << max_threads_per_block << "." << endl;
        exit(EXIT_FAILURE);
    }

    if (!do_consume_data) {
        cout << "We will measure the visibility of the flag value only. "
             << "Setting nblocks and nthreads to 1."
             << endl;
        num_blocks = 1;
        num_threads_per_block = 1;
    }

    ASSERT_EQ(check_gdr_support(dev), true);

    if (galloc_fn == gpu_mem_alloc)
        cout << "gpu alloc fn: hipMalloc" << endl;
    else
        cout << "gpu alloc fn: hipMemCreate" << endl;

    if (do_consume_data) {
        data_buffer_size = PAGE_ROUND_UP(data_size, GPU_PAGE_SIZE);

        ASSERTDRV(galloc_fn(&A_mhandle, data_buffer_size, true, true));
        d_A = A_mhandle.ptr;
        cout << "d_A device ptr: 0x" << hex << d_A << dec << endl;

        ASSERTDRV(galloc_fn(&B_mhandle, data_buffer_size, true, true));
        d_B = B_mhandle.ptr;
        cout << "d_B device ptr: 0x" << hex << d_B << dec << endl;

        ASSERTDRV(hipMemAllocHost((void **)&init_buf, data_size));
        ASSERT_NEQ(init_buf, (void*)0);

        // Just set it to a random value. We don't use the content anyway.
        memset(init_buf, 0xaf, data_size);
    }

    ASSERTDRV(galloc_fn(&gpu_flag_mhandle, sizeof(*g_gpu_flag_buf), true, true));
    d_gpu_flag_buf = gpu_flag_mhandle.ptr;
    cout << "gpu flag device ptr: 0x" << hex << d_gpu_flag_buf << dec << endl;

    ASSERTDRV(hipMemsetD8(d_gpu_flag_buf, 0, sizeof(*g_gpu_flag_buf)));

    ASSERTDRV(hipHostAlloc((void **)&h_cpu_flag_buf, sizeof(*h_cpu_flag_buf) * num_blocks, hipHostMallocPortable | hipHostMallocMapped));
    ASSERT_NEQ(h_cpu_flag_buf, (void*)0);
    ASSERTDRV(hipHostGetDevicePointer(&d_cpu_flag_buf, h_cpu_flag_buf, 0));
    memset(h_cpu_flag_buf, 0, sizeof(*h_cpu_flag_buf) * num_blocks);


    gdr_t g = gdr_open_safe();

    gdr_mh_t gpu_flag_mh;
    void *map_gpu_flag_ptr = NULL;
    gdr_info_t gpu_flag_info;
    int gpu_flag_off;

    gdr_mh_t A_mh;
    void *map_A_ptr = NULL;
    gdr_info_t A_info;
    int A_off;

    BEGIN_CHECK {
        // tokens are optional in CUDA 6.0
        ASSERT_EQ(gdr_pin_buffer(g, d_gpu_flag_buf, sizeof(*g_gpu_flag_buf), 0, 0, &gpu_flag_mh), 0);
        ASSERT_NEQ(gpu_flag_mh, null_mh);

        ASSERT_EQ(gdr_map(g, gpu_flag_mh, &map_gpu_flag_ptr, sizeof(*g_gpu_flag_buf)), 0);
        cout << "map_gpu_flag_ptr: " << map_gpu_flag_ptr << endl;

        ASSERT_EQ(gdr_get_info(g, gpu_flag_mh, &gpu_flag_info), 0);
        cout << "gpu_flag_info.va: " << hex << gpu_flag_info.va << dec << endl;
        cout << "gpu_flag_info.mapped_size: " << gpu_flag_info.mapped_size << endl;
        cout << "gpu_flag_info.page_size: " << gpu_flag_info.page_size << endl;
        cout << "gpu_flag_info.mapped: " << gpu_flag_info.mapped << endl;
        cout << "gpu_flag_info.wc_mapping: " << gpu_flag_info.wc_mapping << endl;

        // remember that mappings start on a 64KB boundary, so let's
        // calculate the offset from the head of the mapping to the
        // beginning of the buffer
        gpu_flag_off = gpu_flag_info.va - d_gpu_flag_buf;
        cout << "gpu_flag page offset: " << gpu_flag_off << endl;

        g_gpu_flag_buf = (uint32_t *)((uintptr_t)map_gpu_flag_ptr + gpu_flag_off);
        cout << "gpu_flag user-space pointer: " << g_gpu_flag_buf << endl;

        if (do_consume_data) {
            ASSERT_EQ(gdr_pin_buffer(g, d_A, data_buffer_size, 0, 0, &A_mh), 0);
            ASSERT_NEQ(A_mh, null_mh);

            ASSERT_EQ(gdr_map(g, A_mh, &map_A_ptr, data_buffer_size), 0);
            cout << "map_A_ptr: " << map_A_ptr << endl;

            ASSERT_EQ(gdr_get_info(g, A_mh, &A_info), 0);
            cout << "A_info.va: " << hex << A_info.va << dec << endl;
            cout << "A_info.mapped_size: " << A_info.mapped_size << endl;
            cout << "A_info.page_size: " << A_info.page_size << endl;
            cout << "A_info.mapped: " << A_info.mapped << endl;
            cout << "A_info.wc_mapping: " << A_info.wc_mapping << endl;

            // remember that mappings start on a 64KB boundary, so let's
            // calculate the offset from the head of the mapping to the
            // beginning of the buffer
            A_off = A_info.va - d_A;
            cout << "A page offset: " << A_off << endl;

            g_A = (uint32_t *)((uintptr_t)map_A_ptr + A_off);
            cout << "A user-space pointer: " << (void *)g_A << endl;

            cout << "Measuring the latency of data + flag model." << endl
                 << "CPU does gdr_copy_to_mapping for data followed by gdr_copy_to_mapping for flag. "
                 << "GPU polls on the flag. Upon observing the upate, it consumes the data. "
                 << "When everything is done, GPU notifies CPU back via hipHostAlloc'd buffer." << endl
                 << "We report the round-trip time from when CPU starts writing the data until it observes the notification from GPU." << endl
                 << endl;

            cout << "Running " << num_iters << " iterations with data size "
                 << data_size << " bytes and flag size " << sizeof(*g_gpu_flag_buf) << " bytes."
                 << endl;

            pp_data_kernel<<< num_blocks, num_threads_per_block >>>((uint32_t *)d_gpu_flag_buf, (uint32_t *)d_cpu_flag_buf, num_iters, (uint32_t *)d_A, (uint32_t *)d_B, data_size);
        }
        else {
            cout << "Measuring the visibility latency of the flag value." << endl
                 << "CPU does gdr_copy_to_mapping, and GPU notifies back via hipHostAlloc'd buffer." << endl
                 << "We report the round-trip time from when CPU updates the flag value until it observes the notification from GPU." << endl
                 << endl;

            cout << "Running " << num_iters << " iterations with flag size " << sizeof(*g_gpu_flag_buf) << " bytes." << endl;

            pp_kernel<<< num_blocks, num_threads_per_block >>>((uint32_t *)d_gpu_flag_buf, (uint32_t *)d_cpu_flag_buf, num_iters);
        }

        // Catching any potential errors. hipErrorNotReady means the kernel
        // is running. We expect to see this status instead of hipSuccess
        // because the kernel must wait for signal from CPU, which occurs after
        // this line.
        ASSERT_EQ(hipStreamQuery(0), hipErrorNotReady);

        uint32_t i = 1;
        uint32_t val;
        unsigned int cpu_flag_idx = 0;
        // Wait for pp_kernel to be ready before starting the time measurement.
        clock_gettime(MYCLOCK, &beg);
        do {
            val = READ_ONCE(h_cpu_flag_buf[cpu_flag_idx]);
            if (val == i)
                ++cpu_flag_idx;
            else
                check_timeout(beg, timeout_us);
        }
        while (cpu_flag_idx < num_blocks);
        LB();

        // Restart the timer for measurement.
        clock_gettime(MYCLOCK, &beg);
        while (i < num_iters) {
            if (do_consume_data) {
                gdr_copy_to_mapping(A_mh, g_A, init_buf, data_size);
                SB();
            }
            gdr_copy_to_mapping(gpu_flag_mh, g_gpu_flag_buf, &val, sizeof(g_gpu_flag_buf));
            SB();

            cpu_flag_idx = 0;
            do {
                val = READ_ONCE(h_cpu_flag_buf[cpu_flag_idx]);
                if (val == i + 1)
                    ++cpu_flag_idx;
                else
                    check_timeout(beg, timeout_us);
            }
            while (cpu_flag_idx < num_blocks);
            LB();
            i = val;
        }
        clock_gettime(MYCLOCK, &end);

        ASSERTDRV(hipStreamSynchronize(0));

        clock_gettime(MYCLOCK, &end);
        lat_us = time_diff(beg, end) / (double)num_iters;

        cout << "Round-trip latency per iteration is " << lat_us << " us" << endl;

        cout << "unmapping buffer" << endl;
        ASSERT_EQ(gdr_unmap(g, gpu_flag_mh, map_gpu_flag_ptr, sizeof(*g_gpu_flag_buf)), 0);
        if (do_consume_data)
            ASSERT_EQ(gdr_unmap(g, A_mh, map_A_ptr, data_buffer_size), 0);

        cout << "unpinning buffer" << endl;
        ASSERT_EQ(gdr_unpin_buffer(g, gpu_flag_mh), 0);
        if (do_consume_data)
            ASSERT_EQ(gdr_unpin_buffer(g, A_mh), 0);
    } END_CHECK;

    cout << "closing gdrdrv" << endl;
    ASSERT_EQ(gdr_close(g), 0);

    ASSERTDRV(hipHostFree(h_cpu_flag_buf));
    ASSERTDRV(gfree_fn(&gpu_flag_mhandle));

    if (do_consume_data) {
        ASSERTDRV(gfree_fn(&A_mhandle));
        ASSERTDRV(gfree_fn(&B_mhandle));
        ASSERTDRV(hipHostFree(init_buf));
    }

    return 0;
}

/*
 * Local variables:
 *  c-indent-level: 4
 *  c-basic-offset: 4
 *  tab-width: 4
 *  indent-tabs-mode: nil
 * End:
 */
